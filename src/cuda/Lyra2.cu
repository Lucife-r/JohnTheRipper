#include "hip/hip_runtime.h"
/**
 * A simple attack against Lyra2 Password Hashing Scheme (PHS).
 * This is a specific implementation, used only to start
 * evaluating GPU attacks. This implementation needs improvement
 * in specific GPU optimization technics.
 *
 * Author: The Lyra PHC team (http://www.lyra2.net/) -- 2015.
 *
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * This file was modified by Agnieszka Bielec <bielecagnieszka8 at gmail.com> on June,2015.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>

#include "Lyra2.h"
#include "Sponge.h"

extern "C" void multPasswordCUDA(unsigned char *K, int kLen, unsigned char *passwords, int pwdLen, unsigned char *salt, int saltLen, unsigned int t_cost, unsigned int m_cost, unsigned int nPARALLEL, unsigned int C_COLS, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize);


/**
 * Generates the passwords for Lyra2 attack.
 *
 * @param t_cost            Parameter to determine the processing time (T)
 * @param m_cost            Memory cost parameter (defines the number of rows of the memory matrix, R)
 * @param totalPasswords    Total number of passwords being tested
 * @param gridSize          GPU grid configuration
 * @param blockSize         GPU block configuration
 * @param printKeys         Defines if the resulting keys will be in the output
 */
__host__ void multPasswordCUDA(unsigned char *K, int kLen, unsigned char *passwords, int pwdLen, unsigned char *salt, int saltLen, unsigned int t_cost, unsigned int m_cost, unsigned int nPARALLEL, unsigned int N_COLS, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize) {
    //=================== Basic variables, with default values =======================//

    //==========================================================================/
    if (m_cost / nPARALLEL < 4) {
        printf("Number of rows too small\n");
        exit(0);
    }

    //Calls the interface to the GPU program
    gpuMult(K, kLen, passwords, pwdLen, salt, saltLen, t_cost, m_cost, nPARALLEL, N_COLS, totalPasswords, gridSize, blockSize);

    hipDeviceReset();
}

int gpuMult(void *K, unsigned int kLen, unsigned char *passwords, unsigned int pwdlen, unsigned char *salt, unsigned int saltlen, unsigned int timeCost, unsigned int nRows, unsigned int nPARALLEL, unsigned int N_COLS, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize) {
    int result = 0;

    //============================= Basic variables ============================//
    int64_t i, j, k; //auxiliary iteration counter

    hipError_t errorCUDA;
    uint64_t sizeSlice = nRows / nPARALLEL;
    //==========================================================================/

    //Checks kernel geometry configuration
    if ((gridSize * blockSize) != (totalPasswords * nPARALLEL)) {
        printf("Error in thread geometry: (gridSize * blockSize) != (totalPasswords * nPARALLEL).\n");
        return -1;
    }

    //========== Initializing the Memory Matrix and Keys =============//
    //Allocates the keys
    unsigned char *pKeys = (unsigned char *) malloc(totalPasswords * nPARALLEL * kLen * sizeof (unsigned char));
    if (pKeys == NULL) {
        return -1;
    }

    // GPU memory matrix alloc:
    // Memory matrix: nRows of N_COLS blocks, each block having BLOCK_LEN_INT64 64-bit words
    uint64_t *memMatrixGPU;
    errorCUDA = hipMalloc((void**) &memMatrixGPU, totalPasswords * nRows * ROW_LEN_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the GPU keys
    unsigned char *pkeysGPU;
    errorCUDA = hipMalloc((void**) &pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof (unsigned char));
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Sponge state: 16 uint64_t, BLOCK_LEN_INT64 words of them for the bitrate (b) and the remainder for the capacity (c)
    uint64_t *stateThreadGPU;
    errorCUDA = hipMalloc((void**) &stateThreadGPU, totalPasswords * nPARALLEL * STATESIZE_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // stateThreadGPU cleanup:
    hipMemset(stateThreadGPU, 0, totalPasswords * nPARALLEL * STATESIZE_BYTES);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        return -2;
    }

    //Allocates the State Index to be absorbed by each thread.
    uint64_t *stateIdxGPU;
    errorCUDA = hipMalloc((void**) &stateIdxGPU, totalPasswords * nPARALLEL * BLOCK_LEN_BLAKE2_SAFE_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the Password in GPU.
    unsigned char *pwdGPU;
    errorCUDA = hipMalloc((void**) &pwdGPU, totalPasswords * pwdlen);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Transfers the password to GPU.
    errorCUDA = hipMemcpy(pwdGPU, passwords, totalPasswords * pwdlen, hipMemcpyHostToDevice);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the Salt in GPU.
    unsigned char *saltGPU;
    errorCUDA = hipMalloc((void**) &saltGPU, saltlen);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Transfers the salt to GPU.
    errorCUDA = hipMemcpy(saltGPU, salt, saltlen, hipMemcpyHostToDevice);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    bootStrapGPU <<<gridSize, blockSize>>>(memMatrixGPU, pkeysGPU, kLen, pwdGPU, pwdlen, saltGPU, saltlen, timeCost, nRows, N_COLS, totalPasswords, nPARALLEL, N_COLS);

    // Needs to wait all threads:
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //============== Initializing the Sponge State =============/
    initState <<<gridSize, blockSize>>>(stateThreadGPU, totalPasswords, nPARALLEL);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //============= Absorbing the input data with the sponge ===============//
    absorbInput <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, stateIdxGPU, pwdGPU, pwdlen, saltlen, totalPasswords, nPARALLEL);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //================================ Setup and Wandering Phase =============================//
    //Initializes M[0]
    reducedSqueezeRow0 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords, nPARALLEL, N_COLS); //The locally copied password is most likely overwritten here

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Initializes M[1]
    reducedDuplexRow1and2 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords, 0, 1, nPARALLEL, N_COLS);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Initializes M[2]
    reducedDuplexRow1and2 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords, 1, 2, nPARALLEL, N_COLS);

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }
if (nPARALLEL == 1)
    // Runs Setup and Wandering Phase
    setupPhaseWanderingGPU_P1 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, sizeSlice, totalPasswords, timeCost, nPARALLEL, N_COLS);

if (nPARALLEL > 1)
    // Runs Setup and Wandering Phase
    setupPhaseWanderingGPU <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, sizeSlice, totalPasswords, timeCost, nPARALLEL, N_COLS);

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error after SetupWandering: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Squeezes the keys
    squeeze <<<gridSize, blockSize>>>(stateThreadGPU, pkeysGPU, kLen, totalPasswords, nPARALLEL);

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Getting the keys back.
    errorCUDA = hipMemcpy(pKeys, pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof (unsigned char), hipMemcpyDeviceToHost);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }


if (nPARALLEL > 1)
{
    // XORs all Keys
    for (k = 0; k < totalPasswords; k++) {
        for (i = 1; i < nPARALLEL; i++) {
            for (j = 0; j < kLen; j++) {
                pKeys[k * kLen * nPARALLEL + j] ^= pKeys[k * kLen * nPARALLEL + i * kLen + j];
            }
        }
    }

    //Move the keys to proper place
    for (k = 1; k < totalPasswords; k++) {
        for (j = 0; j < kLen; j++) {
            pKeys[k * kLen + j] = pKeys[k * kLen * nPARALLEL + j];
        }
    }
}
    // Returns in the correct variable
    memcpy(K, pKeys, totalPasswords * kLen * sizeof (unsigned char));

    //========== Frees the Memory Matrix and Keys =============//
    hipFree(memMatrixGPU);
    hipFree(pkeysGPU);
    hipFree(stateThreadGPU);
    hipFree(stateIdxGPU);
    hipFree(saltGPU);
    hipFree(pwdGPU);

    //Free allKeys
    free(pKeys);
    pKeys = NULL;

    return result;
}
